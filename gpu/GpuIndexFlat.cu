#include "hip/hip_runtime.h"
/*
 * Author: VincentLee
 * Email:  lichlee@yeah.net
 * Created on 2019/01/08.
*/

#include "GpuIndexFlat.h"
#include "../IndexFlat.h"
#include "utils/DeviceUtils.h"
#include "utils/Float16.cuh"
#include "impl/FlatIndex.cuh"

#include <limits>

namespace faiss_v { namespace gpu {

/// Default CPU search size for which we use paged copies
constexpr size_t kMinPageSize = (size_t) 256 * 1024 * 1024;

/// Size above which we page copies from the CPU to GPU (non-paged
/// memory usage)
constexpr size_t kNonPinnedPageSize = (size_t) 256 * 1024 * 1024;

GpuIndexFlat::GpuIndexFlat(GpuResources* resources,
                           const faiss_v::IndexFlat *index,
                           GpuIndexFlatConfig config) :
        GpuIndex(resources, index->d, index->metric_type, config),
        minPagedSize_(kMinPageSize),
        config_(config),
        data_(nullptr) {
    verifySettings_();

    // Flat index doesn't need training
    this->is_trained = true;

    copyFrom(index);
}

GpuIndexFlat::GpuIndexFlat(GpuResources *resources,
                           int dims,
                           MetricType metric,
                           GpuIndexFlatConfig config) :
        GpuIndex(resources, dims, metric, config),
        minPagedSize_(kMinPageSize),
        config_(config),
        data_(nullptr) {
    verifySettings_();

    // Flat index doesn't need training
    this->is_trained = true;

    // Construct index
    DeviceScope scope(device_);
    data_ = new FlatIndex(resources,
                          dims,
                          metric == faiss_v::METRIC_L2,
                          config_.useFlat16,
                          config_.useFloat16Accumulator,
                          config_.storeTransposed,
                          memorySpace_);
}

GpuIndexFlat::~GpuIndexFlat() {
    delete data_;
}

void
GpuIndexFlat::copyFrom(const IndexFlat *index) {
    DeviceScope scope(device_);

    this->d = index->d;
    this->metric_type = index->metric_type;

    // GPU code has 32 bit indices
    FAISSV_THROW_IF_NOT_FMT(index->ntotal <= (Index::idx_t) std::numeric_limits<int>::max(),
                            "GPU index only supports up to %zu indices; "
                                "attempting to copy CPU index with %zu parameters",
                            (size_t) std::numeric_limits<int>::max(),
                            (size_t) index->ntotal);
    this->ntotal = index->ntotal;

    delete data_;
    data_ = new FlatIndex(resources_,
                          this->d,
                          index->metric_type == faiss_v::METRIC_L2,
                          config_.useFlat16,
                          config_.useFloat16Accumulator,
                          config_.storeTransposed,
                          memorySpace_);

    // The index could be empty
    if(index->ntotal > 0) {
        data_->
    }
}

void
GpuIndexFlat::verifySettings_() const {
    // Ensure Hgemm is supported on this device
    if(config_.useFloat16Accumulator) {
#define FAISS_USE_FLOAT16
        FAISSV_THROW_IF_NOT_MSG(config_.useFlat16,
                                "useFloat16Accumulator can only be enabled "
                                    "with useFloat16");
        FAISSV_THROW_IF_NOT_FMT(getDeviceSupportsFloat16Math(config_.device),
                                "Device %d does not support Hgemm "
                                    "(useFloat16Accumulator)",
                                config_.device);
    }
#else
    FAISSV_THROW_IF_NOT_MSG(false, "not compiled with float16 support");
#endif
}

}}